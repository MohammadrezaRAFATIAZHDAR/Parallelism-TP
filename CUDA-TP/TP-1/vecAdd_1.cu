void vecAdd(int *h_in, int *h_out, int n){
    
    int dimgrid = 1;
    int dimblock = 1024;
    int *d_in;
    int *d_out;
    int size = n * sizeof(int)

    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    
    vecSumKernel1<<<dimgrid, dimblock>>>(d_in, d_out, n);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

--global--
void vecSumKernel1(int *d_in, int d_out, int n){
    int idx = threadIdx.x;

    if (idx < n){
      
        d_out[idx] += d_in[idx];
        
    }
}